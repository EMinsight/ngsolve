
#include <hip/hip_runtime.h>
// x = val
__global__ void SetScalarKernel (double val, int n, double * x)
{
  int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockDim.x)
    x[i] = val;
}

void SetScalar (double val, int n, double * x)
{
  SetScalarKernel<<<1,128>>> (val, n, x);
} 


// y = D * x
__global__ void MultDiagonalKernel (int n, double * D, double * x, double * y)
{
  int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockDim.x)
    y[i] = D[i] * x[i];
}

void MultDiagonal (int n, double * D, double * x, double * y)
{
  MultDiagonalKernel<<<1,128>>> (n, D, x, y);
} 


// y += alpha D * x
__global__ void MultAddDiagonalKernel (int n, double alpha, double * D, double * x, double * y)
{
  int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockDim.x)
    y[i] += alpha * D[i] * x[i];
}

void MultAddDiagonal (int n, double alpha, double * D, double * x, double * y)
{
  MultAddDiagonalKernel<<<1,128>>> (n, alpha, D, x, y);
} 




/* ************** kernels for ConstantEBE Matrix ********************** */


__global__ void ConstEBEKernelCopyInKernel (int numblocks, int bs, int * row_dnums, double * dev_ux, double * dev_hx)
{
  int tid = threadIdx.x;
  
  for (int r = 0; r < numblocks; r++)
    {
      for (int i = tid; i < bs; i += blockDim.x)
        dev_hx[r*bs+i] = dev_ux[row_dnums[r*bs+i]];
    }
}

void ConstEBEKernelCopyIn (int numblocks, int bs, int * row_dnums, double * dev_ux, double * dev_hx)
{
  ConstEBEKernelCopyInKernel<<<1,128>>> (numblocks, bs, row_dnums, dev_ux, dev_hx);
}

__global__ void ConstEBEKernelCopyOutKernel (int numblocks, int bs, int *  col_dnums, double * dev_hy, double * dev_uy)
{
  int tid = threadIdx.x;
  
  for (int r = 0; r < numblocks; r++)
    {
      for (int i = tid; i < bs; i += blockDim.x)
        dev_uy[col_dnums[r*bs+i]] += dev_hy[r*bs+i];
    }
}

void ConstEBEKernelCopyOut (int numblocks, int bs, int * col_dnums, double * dev_hy, double * dev_uy)
{
  ConstEBEKernelCopyOutKernel<<<1,128>>> (numblocks, bs, col_dnums, dev_hy, dev_uy);
}

